#include "hip/hip_runtime.h"
/*!
  \file gpp_cuda_math.cu
  \rst
  This file contains implementations of all GPU functions. There are both device code (executed on
  GPU device) and host code (executed on CPU), and they are compiled by NVCC, which is a NVIDIA CUDA
  compiler.
\endrst*/

#include "gpp_cuda_math.hpp"

#include <stdint.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include <algorithm>

/*!\rst
  Macro to stringify the expansion of a macro. For example, say we are on line 53:

  * ``#__LINE__ --> "__LINE__"``
  * ``OL_CUDA_STRINGIFY_EXPANSION(__LINE__) --> "53"``

  ``OL_CUDA_STRINGIFY_EXPANSION_INNER`` is not meant to be used directly;
  but we need ``#x`` in a macro for this expansion to work.

  This is a standard trick; see bottom of:
  http://gcc.gnu.org/onlinedocs/cpp/Stringification.html
\endrst*/
#define OL_CUDA_STRINGIFY_EXPANSION_INNER(x) #x
#define OL_CUDA_STRINGIFY_EXPANSION(x) OL_CUDA_STRINGIFY_EXPANSION_INNER(x)

/*!\rst
  Macro to stringify and format the current file and line number. For
  example, if the macro is invoked from line 893 of file gpp_foo.cpp,
  this macro produces the compile-time string-constant:
  ``(gpp_foo.cpp: 893)``
\endrst*/
#define OL_CUDA_STRINGIFY_FILE_AND_LINE "(" __FILE__ ": " OL_CUDA_STRINGIFY_EXPANSION(__LINE__) ")"

/*!\rst
  Macro that checks error message (with type hipError_t) returned by CUDA API functions, and if there is error occurred,
  the macro produces a C struct containing error message, function name where error occured, file name and line info, and 
  then terminate the function.
\endrst*/
#define OL_CUDA_ERROR_RETURN(X) do {hipError_t _error_code = (X); if (_error_code != hipSuccess) {CudaError _err = {_error_code, OL_CUDA_STRINGIFY_FILE_AND_LINE, __func__}; return _err;}} while (0)

namespace optimal_learning {

namespace {  // functions run on gpu device
/*!\rst
  Special case of GeneralMatrixVectorMultiply.  As long as A has zeros in the strict upper-triangle,
  GeneralMatrixVectorMultiply will work too (but take ``>= 2x`` as long).

  Computes results IN-PLACE.
  Avoids accessing the strict upper triangle of A.

  Should be equivalent to BLAS call:
  ``dtrmv('L', trans, 'N', size_m, A, size_m, x, 1);``
\endrst*/
__device__ void CudaTriangularMatrixVectorMultiply(double const * __restrict__ A, int size_m, double * __restrict__ x) {
  double temp;
  A += size_m * (size_m-1);
  for (int j = size_m-1; j >= 0; --j) {  // i.e., j >= 0
    temp = x[j];
    for (int i = size_m-1; i >= j+1; --i) {
      // handles sub-diagonal contributions from j-th column
      x[i] += temp*A[i];
    }
    x[j] *= A[j];  // handles j-th on-diagonal component
    A -= size_m;
  }
}

/*!\rst
  This is reduced version of GeneralMatrixVectorMultiply(...) in gpp_linear_algebra.cpp, and this function computes
  y = y - A * x (aka alpha = -1.0, beta = 1.0)
\endrst*/
__device__ void CudaGeneralMatrixVectorMultiply(double const * __restrict__ A, double const * __restrict__ x, int size_m, int size_n, int lda, double * __restrict__ y) {
  double temp;
  for (int i = 0; i < size_n; ++i) {
    temp = -1.0 * x[i];
    for (int j = 0; j < size_m; ++j) {
      y[j] += A[j]*temp;
    }
    A += lda;
  }
}

/*!\rst
  This inline function copies [begin, begin+1, ..., end-1] elements from one array to the other, if bound < end, then end = bound
\endrst*/
__forceinline__ __device__ void CudaCopyElements(int begin, int end, int bound, double const * __restrict__ origin, double * __restrict__ destination) {
    int local_end = end < bound ? end : bound;
    for (int idx = begin; idx < local_end; ++idx) {
        destination[idx] = origin[idx];
    }
}

/*!\rst
  Device code to compute Expected Improvement by Monte-Carlo on GPU

  \param
    :mu[num_union]: the mean of the GP evaluated at points interested
    :chol_var[num_union][num_union]: cholesky factorization of the GP variance evaluated at points interested
    :num_union: number of the points interested
    :num_iteration: number of iterations performed on each thread for MC evaluation
    :best: best function evaluation obtained so far
    :seed: seed for RNG
    :ei_storage[num_threads][num_blocks]: array storing values of EI on GPU
    :gpu_random_number_ei[num_union][num_iteration][num_threads][num_blocks]: array storing random
      numbers used for computing EI, for testing purpose only
    :configure_for_test: whether record random_number_ei or not
  \output
    :ei_storage[num_threads][num_blocks]: each thread write result of computed EI to its corresponding position
    :gpu_random_number_ei[num_union][num_iteration][num_threads][num_blocks]: write random numbers
      used for computing EI into the array, for testing purpose only
\endrst*/
__global__ void CudaComputeEIGpu(double const * __restrict__ mu, double const * __restrict__ chol_var,
                                 int num_union, int num_iteration, double best, uint64_t seed,
                                 double * __restrict__ ei_storage, double* __restrict__ gpu_random_number_ei,
                                 bool configure_for_test) {
  // copy mu, chol_var to shared memory mu_local & chol_var_local
  // For multiple dynamically sized arrays in a single kernel, declare a single extern unsized array, and use
  // pointers into it to divide it into multiple arrays
  // refer to http://devblogs.nvidia.com/parallelforall/using-shared-memory-cuda-cc/
  extern __shared__ double storage[];
  double * chol_var_local = storage;
  double * mu_local = chol_var_local + num_union * num_union;
  const int idx = threadIdx.x;
  const int IDX = threadIdx.x + blockDim.x * blockIdx.x;
  int chunk_size = (num_union * num_union - 1)/ blockDim.x + 1;
  CudaCopyElements(chunk_size * idx, chunk_size * (idx + 1), num_union * num_union, chol_var, chol_var_local);
  chunk_size = (num_union - 1)/ blockDim.x + 1;
  CudaCopyElements(chunk_size * idx, chunk_size * (idx + 1), num_union,  mu, mu_local);
  __syncthreads();

  // MC start
  // RNG setup
  uint64_t local_seed = seed + IDX;
  hiprandState random_state;
  // seed a random number generator
  hiprand_init(local_seed, 0, 0, &random_state);

  double *normals = reinterpret_cast<double *>(malloc(sizeof(*chol_var_local) * num_union));
  double agg = 0.0;
  double improvement_this_step;
  double EI;

  for (int mc = 0; mc < num_iteration; ++mc) {
    improvement_this_step = 0.0;
    for (int i = 0; i < num_union; ++i) {
        normals[i] = hiprand_normal_double(&random_state);
        // If configure_for_test is true, random numbers used in MC computations will be saved as output.
        // In fact we will let EI compuation on CPU use the same sequence of random numbers saved here,
        // so that EI compuation on CPU & GPU can be compared directly for unit test purpose.
        if (configure_for_test) {
            gpu_random_number_ei[IDX * num_iteration * num_union + mc * num_union + i] = normals[i];
        }
    }
    CudaTriangularMatrixVectorMultiply(chol_var_local, num_union, normals);
    for (int i = 0; i < num_union; ++i) {
        EI = best - (mu_local[i] + normals[i]);
        improvement_this_step = fmax(EI, improvement_this_step);
    }
    agg += improvement_this_step;
  }
  ei_storage[IDX] = agg / static_cast<double>(num_iteration);
  free(normals);
}

/*!\rst
  Device code to compute Gradient of Expected Improvement by Monte-Carlo on GPU

  \param
    :mu[num_union]: the mean of the GP evaluated at points interested
    :chol_var[num_union][num_union]: cholesky factorization of the GP variance evaluated at points interested
    :grad_mu[dim][num_to_sample]: the gradient of mean of the GP evaluated at points interested
    :grad_chol_var[dim][num_union][num_union][num_to_sample]: gradient of cholesky factorization of the GP variance
      evaluated at points interested
    :num_union: number of the union of points (aka q+p)
    :num_to_sample: number of points to sample (aka q)
    :dim: dimension of point space
    :num_iteration: number of iterations performed on each thread for MC evaluation
    :best: best function evaluation obtained so far
    :seed: seed for RNG
    :grad_ei_storage[dim][num_to_sample][num_threads][num_blocks]: A vector storing result of grad_ei from each thread
    :gpu_random_number_grad_ei[num_union][num_itreration][num_threads][num_blocks]: array storing
      random numbers used for computing gradEI, for testing purpose only
    :configure_for_test: whether record random_number_grad_ei or not
  \output
    :grad_ei_storage[dim][num_to_sample][num_threads][num_blocks]: each thread write result of grad_ei
      to its corresponding positions
    :gpu_random_number_grad_ei[num_union][num_iteration][num_threads][num_blocks]: write random numbers
      used for computing gradEI to the array, for testing purpose only
\endrst*/
__global__ void CudaComputeGradEIGpu(double const * __restrict__ mu, double const * __restrict__ chol_var,
                                     double const * __restrict__ grad_mu, double const * __restrict__ grad_chol_var,
                                     int num_union, int num_to_sample, int dim, int num_iteration, double best,
                                     uint64_t seed,  double * __restrict__ grad_ei_storage,
                                     double* __restrict__ gpu_random_number_grad_ei, bool configure_for_test) {
  // copy mu, chol_var, grad_mu, grad_chol_var to shared memory
  extern __shared__ double storage[];
  double * mu_local = storage;
  double * chol_var_local = mu_local + num_union;
  double * grad_mu_local = chol_var_local + num_union * num_union;
  double * grad_chol_var_local = grad_mu_local + num_to_sample * dim;
  const int idx = threadIdx.x;
  const int IDX = threadIdx.x + blockDim.x * blockIdx.x;
  int chunk_size = (num_to_sample * num_union * num_union * dim - 1)/ blockDim.x + 1;
  CudaCopyElements(chunk_size * idx, chunk_size * (idx + 1), num_to_sample * num_union * num_union * dim,
                  grad_chol_var, grad_chol_var_local);
  chunk_size = (num_union * num_union - 1)/ blockDim.x + 1;
  CudaCopyElements(chunk_size * idx, chunk_size * (idx + 1), num_union * num_union, chol_var, chol_var_local);
  chunk_size = (num_to_sample * dim - 1)/ blockDim.x + 1;
  CudaCopyElements(chunk_size * idx, chunk_size * (idx + 1), num_to_sample * dim, grad_mu, grad_mu_local);
  chunk_size = (num_union - 1)/ blockDim.x + 1;
  CudaCopyElements(chunk_size * idx, chunk_size * (idx + 1), num_union, mu, mu_local);
  __syncthreads();

  int i, k, mc, winner;
  double EI, improvement_this_step;
  // RNG setup
  uint64_t local_seed = seed + IDX;
  hiprandState random_state;
  hiprand_init(local_seed, 0, 0, &random_state);
  double* normals = reinterpret_cast<double*>(malloc(sizeof(*mu_local) * num_union));
  double* normals_copy = reinterpret_cast<double*>(malloc(sizeof(*mu_local) * num_union));
  // initialize grad_ei_storage
  for (int i = 0; i < (num_to_sample * dim); ++i) {
      grad_ei_storage[IDX*num_to_sample*dim + i] = 0.0;
  }
  // MC step start
  for (mc = 0; mc < num_iteration; ++mc) {
      improvement_this_step = 0.0;
      winner = -1;
      for (i = 0; i < num_union; ++i) {
          normals[i] = hiprand_normal_double(&random_state);
          normals_copy[i] = normals[i];
            // If configure_for_test is true, random numbers used in MC computations will be saved as output.
            // In fact we will let grad_ei compuation on CPU use the same sequence of random numbers saved here,
            // so that grad_ei compuation on CPU & GPU can be compared directly for unit test purpose.
          if (configure_for_test) {
              gpu_random_number_grad_ei[IDX * num_iteration * num_union + mc * num_union + i] = normals[i];
          }
      }
      CudaTriangularMatrixVectorMultiply(chol_var_local, num_union, normals);
      for (i = 0; i < num_union; ++i) {
          EI = best - (mu_local[i] + normals[i]);
          if (EI > improvement_this_step) {
              improvement_this_step = EI;
              winner = i;
          }
      }
      if (improvement_this_step > 0.0) {
          if (winner < num_to_sample) {
              for (k = 0; k < dim; ++k) {
                  grad_ei_storage[IDX*num_to_sample*dim + winner * dim + k] -= grad_mu_local[winner * dim + k];
              }
          }
          for (i = 0; i < num_to_sample; ++i) {   // derivative w.r.t ith point
              CudaGeneralMatrixVectorMultiply(grad_chol_var_local + i * num_union * num_union * dim +
                                              winner * num_union * dim, normals_copy, dim, num_union,
                                              dim, grad_ei_storage + IDX * num_to_sample * dim + i * dim);
          }
      }
  }

  for (int i = 0; i < num_to_sample*dim; ++i) {
      grad_ei_storage[IDX*num_to_sample*dim + i] /= static_cast<double>(num_iteration);
  }
  free(normals);
  free(normals_copy);
}

}  // end unnamed namespace

CudaError CudaAllocateMemForDoubleVector(int num_doubles, double** __restrict__ address_of_ptr_to_gpu_memory) {
  CudaError _success = {hipSuccess, OL_CUDA_STRINGIFY_FILE_AND_LINE, __func__};
  int mem_size = num_doubles * sizeof(**address_of_ptr_to_gpu_memory);
  OL_CUDA_ERROR_RETURN(hipMalloc(reinterpret_cast<void**>(address_of_ptr_to_gpu_memory), mem_size));
  return _success;
}

void CudaFreeMem(double* __restrict__ ptr_to_gpu_memory) {
  hipFree(ptr_to_gpu_memory);
}

CudaError CudaGetEI(double * __restrict__ mu, double * __restrict__ chol_var, int num_union, int num_mc,
                    uint64_t seed, double best, bool configure_for_test, double * __restrict__ random_number_ei,
                    double * __restrict__ ei_val, double * __restrict__ gpu_mu, double * __restrict__ gpu_chol_var,
                    double* __restrict__ gpu_random_number_ei, double * __restrict__ gpu_ei_storage) {
  *ei_val = 0.0;
  CudaError _success = {hipSuccess, OL_CUDA_STRINGIFY_FILE_AND_LINE, __func__};

  // We assign kEINumBlocks blocks and kEINumThreads threads/block for EI computation, so there are
  // (kEINumBlocks * kEINumThreads) threads in total to execute kernel function in parallel
  dim3 threads(kEINumThreads);
  dim3 grid(kEINumBlocks);
  double ei_storage[kEINumThreads * kEINumBlocks];
  int num_iteration = num_mc / (kEINumThreads * kEINumBlocks) + 1;   // make sure num_iteration is always >= 1

  int mem_size_mu = num_union * sizeof(*mu);
  int mem_size_chol_var = num_union * num_union * sizeof(*mu);
  int mem_size_ei_storage = kEINumThreads * kEINumBlocks * sizeof(*mu);
  // copy mu, chol_var to GPU
  OL_CUDA_ERROR_RETURN(hipMemcpy(gpu_mu, mu, mem_size_mu, hipMemcpyHostToDevice));
  OL_CUDA_ERROR_RETURN(hipMemcpy(gpu_chol_var, chol_var, mem_size_chol_var, hipMemcpyHostToDevice));
  // execute kernel
  CudaComputeEIGpu <<< grid, threads, num_union*sizeof(*mu)+num_union*num_union*sizeof(*mu) >>>
                   (gpu_mu, gpu_chol_var, num_union, num_iteration, best, seed, gpu_ei_storage,
                    gpu_random_number_ei, configure_for_test);
  OL_CUDA_ERROR_RETURN(hipPeekAtLastError());
  // copy gpu_ei_storage back to CPU
  OL_CUDA_ERROR_RETURN(hipMemcpy(ei_storage, gpu_ei_storage, mem_size_ei_storage, hipMemcpyDeviceToHost));
  // copy gpu_random_number_ei back to CPU if configure_for_test is on
  if (configure_for_test) {
      int mem_size_random_number_ei = num_iteration * kEINumThreads * kEINumBlocks * num_union * sizeof(*mu);
      OL_CUDA_ERROR_RETURN(hipMemcpy(random_number_ei, gpu_random_number_ei, mem_size_random_number_ei, hipMemcpyDeviceToHost));
  }
  // average ei_storage
  double ave = 0.0;
  for (int i = 0; i < (kEINumThreads*kEINumBlocks); ++i) {
      ave += ei_storage[i];
  }
  *ei_val = ave / static_cast<double>(kEINumThreads*kEINumBlocks);
  return _success;
}

CudaError CudaGetGradEI(double * __restrict__ mu, double * __restrict__ chol_var, double * __restrict__ grad_mu,
                        double * __restrict__ grad_chol_var, int num_union, int num_to_sample, int dim, int num_mc,
                        uint64_t seed, double best, bool configure_for_test, double* __restrict__ random_number_grad_ei,
                        double * __restrict__ grad_ei, double * __restrict__ gpu_mu, double * __restrict__ gpu_chol_var,
                        double * __restrict__ gpu_grad_mu, double * __restrict__ gpu_grad_chol_var,
                        double* __restrict__ gpu_random_number_grad_ei, double * __restrict__ gpu_grad_ei_storage) {
  CudaError _success = {hipSuccess, OL_CUDA_STRINGIFY_FILE_AND_LINE, __func__};

  double grad_ei_storage[num_to_sample * dim * kGradEINumThreads * kGradEINumBlocks];
  std::fill(grad_ei, grad_ei + num_to_sample * dim, 0.0);

  // We assign kGradEINumBlocks blocks and kGradEINumThreads threads/block for grad_ei computation,
  // so there are (kGradEINumBlocks * kGradEINumThreads) threads in total to execute kernel function
  // in parallel
  dim3 threads(kGradEINumThreads);
  dim3 grid(kGradEINumBlocks);
  int num_iteration = num_mc / (kGradEINumThreads * kGradEINumBlocks) + 1;   // make sure num_iteration is always >= 1

  int mem_size_mu = num_union * sizeof(*mu);
  int mem_size_grad_mu = num_to_sample * dim * sizeof(*mu);
  int mem_size_chol_var = num_union * num_union *sizeof(*mu);
  int mem_size_grad_chol_var = num_to_sample * num_union * num_union * dim * sizeof(*mu);
  int mem_size_grad_ei_storage= kGradEINumThreads * kGradEINumBlocks * num_to_sample * dim * sizeof(*mu);

  OL_CUDA_ERROR_RETURN(hipMemcpy(gpu_mu, mu, mem_size_mu, hipMemcpyHostToDevice));
  OL_CUDA_ERROR_RETURN(hipMemcpy(gpu_grad_mu, grad_mu, mem_size_grad_mu, hipMemcpyHostToDevice));
  OL_CUDA_ERROR_RETURN(hipMemcpy(gpu_chol_var, chol_var, mem_size_chol_var, hipMemcpyHostToDevice));
  OL_CUDA_ERROR_RETURN(hipMemcpy(gpu_grad_chol_var, grad_chol_var, mem_size_grad_chol_var, hipMemcpyHostToDevice));

  // execute kernel
  // inputs: gpu_mu, gpu_chol_var, gpu_grad_mu, gpu_grad_chol_var, best, num_union, num_to_sample, dim, num_iteration, seed
  // output: gpu_grad_ei_storage
  CudaComputeGradEIGpu <<< grid, threads, mem_size_mu+mem_size_chol_var+mem_size_grad_mu+mem_size_grad_chol_var >>>
                       (gpu_mu, gpu_chol_var, gpu_grad_mu, gpu_grad_chol_var, num_union, num_to_sample, dim,
                        num_iteration, best, seed, gpu_grad_ei_storage, gpu_random_number_grad_ei, configure_for_test);
  OL_CUDA_ERROR_RETURN(hipPeekAtLastError());

  OL_CUDA_ERROR_RETURN(hipMemcpy(grad_ei_storage, gpu_grad_ei_storage, mem_size_grad_ei_storage, hipMemcpyDeviceToHost));
  // copy gpu_random_number_grad_ei back to CPU if configure_for_test is on
  if (configure_for_test) {
      int mem_size_random_number_grad_ei = num_iteration * kGradEINumThreads * kGradEINumBlocks * num_union * sizeof(*mu);
      OL_CUDA_ERROR_RETURN(hipMemcpy(random_number_grad_ei, gpu_random_number_grad_ei, mem_size_random_number_grad_ei, hipMemcpyDeviceToHost));
  }

  // The code block below extracts grad_ei from grad_ei_storage, which is output from the function
  // "CudaGetGradEI" run on gpu. The way to do that is for each component of grad_ei, we find all
  // the threads calculating the corresponding component and average over the threads.
  for (int n = 0; n < (kGradEINumThreads*kGradEINumBlocks); ++n) {
      for (int i = 0; i < num_to_sample*dim; ++i) {
          grad_ei[i] += grad_ei_storage[n*num_to_sample*dim + i];
      }
  }
  for (int i = 0; i < num_to_sample*dim; ++i) {
      grad_ei[i] /= static_cast<double>(kGradEINumThreads*kGradEINumBlocks);
  }
  return _success;
}

CudaError CudaSetDevice(int devID) {
  CudaError _success = {hipSuccess, OL_CUDA_STRINGIFY_FILE_AND_LINE, __func__};
  OL_CUDA_ERROR_RETURN(hipSetDevice(devID));
  return _success;
}

}    // end namespace optimal_learning
